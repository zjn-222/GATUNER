#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
//全Fp64
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <math.h>
#include <stdio.h>

//测量时间加的
#include <chrono>
#include <iostream>
#include <fstream>
#include <iostream>
#include <iomanip>
//json库
#include "/usr/include/nlohmann/json.hpp"
using json = nlohmann::json;

#define N 1000000
// 修改为双精度常量
#define PI 3.14159265358979323846  // 移除L后缀
#define ANS 5.795776322412856      // 移除L后缀

// __device__ double fun(double x);
__device__ double fun(double x2) {
    double d2 = 1.0;
    double t3 = x2;
    
    for (int k = 1; k <= 5; ++k) {
        d2 =d2* 2.0;
        t3 =t3+ sin(d2 * x2) / d2;
    }
    return t3;
}

__global__ void compute_t(double* d_t, int num_points, double h1) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_points) {
        double x1;
        x1 = i * h1;
        d_t[i] = fun(x1);
    }
}

__global__ void compute_and_sum(const double* d_t, int num_segments, double h2, double* d_sum) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_segments) {
        double dt ;
        dt = d_t[i+1] - d_t[i];
        atomicAdd(d_sum, sqrt(h2*h2 + dt*dt));
    }
}



int main() {



    double h = PI / N;
    const int num_points = N + 1;
    const int num_segments = N;

    // 分配设备内存
    double *d_t = nullptr, *d_sum = nullptr;
    hipMalloc(&d_t, num_points * sizeof(double));
    hipMalloc(&d_sum, sizeof(double));
    hipMemset(d_sum, 0, sizeof(double));

  // 获取开始时间
  auto start = std::chrono::high_resolution_clock::now();

    // 计算函数值
    const int block_size = 256;
    int grid_size = (num_points + block_size - 1) / block_size;
    compute_t<<<grid_size, block_size>>>(d_t, num_points, h);
    hipDeviceSynchronize();

    // 计算并累加线段长度
    grid_size = (num_segments + block_size - 1) / block_size;
    compute_and_sum<<<grid_size, block_size>>>(d_t, num_segments, h, d_sum);
    hipDeviceSynchronize();

    // 获取结果
    double s1 = 0;
    hipMemcpy(&s1, d_sum, sizeof(double), hipMemcpyDeviceToHost);

  // 获取结束时间
  auto end = std::chrono::high_resolution_clock::now();

    // 修正printf格式说明符
    printf("Calculated value: %.15f\n", s1);
    printf("Expected value:    %.15f\n", ANS);  // ANS现在为双精度
    printf("Absolute error:    %e\n", fabs(ANS - s1));  // 结果转为双精度

    // 释放资源
    hipFree(d_t);
    hipFree(d_sum);


  // 计算并输出执行时间
  auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
  std::cout << "Execution time: " << duration.count() << " nanoseconds" << std::endl;
  double program_error=abs(fabs(ANS - s1));
  printf("绝对误差为：%.15e\n",program_error);
  printf("结果为：%.15e\n",s1);
  //并行版
  //每个chromosome个体独立写入数据到/home/zjn/mytool4/josn/chromosome1.json，最终再合并成RuntimeAndError.json
  json data;
  // 更新当前染色体的数据
  data["chromosome1"]["runtime"] = duration.count(); // 此处键名会被 sed 动态替换
  data["chromosome1"]["error"] = program_error;
  std::string json_path="/home/zjn/mytool4/json/";
  json_path+="chromosome1";
  json_path+=".json";
  // 写回文件
  std::ofstream file(json_path);
  std::cout << "RuntimeAndError.json更新完成"<< std::endl;
  if (file.is_open()) {
      file << data.dump(4);
      file.close();
  } else {
      std::cerr << "无法写入文件！" << std::endl;
      return 1;
  }
    return 0;
}