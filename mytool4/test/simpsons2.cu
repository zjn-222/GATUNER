#include "hip/hip_runtime.h"
//全Fp64
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <math.h>
#include <stdio.h>


//测量时间加的
#include <chrono>
#include <iostream>
#include <fstream>
#include <iostream>
#include <iomanip>
//json库
#include "/usr/include/nlohmann/json.hpp"
using json = nlohmann::json;

// typedef half fp16; // CUDA半精度类型
// typedef float fp32;
// typedef double fp64;

// 设备端全局变量
__device__ double pi; // 声明在GPU上的浮点变量
// 设备函数
__device__ double fun(double xarg) {  // GPU上的计算函数
    // return __float2half(sinf(pi * xarg));
    return sin(pi * xarg);
}


__global__ void compute_pi_kernel() {
    pi = acos(-1.0f); // 直接在GPU上计算π
}

// 核函数
__global__ void integrate_kernel(double a, double b, double* s1, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 2 * n) return;

    double h1;
    h1 = (b - a) / (2.0f * n);
    double x ;
    x=  idx * h1;

    double val;
    if (idx % 2 == 0) {
        // val = __float2half(4.0f) * fun(x);
        val = (4.0f) * (fun(x));
    } else {
        // val = __float2half(2.0f) * fun(x);
        val = (4.0f) * (fun(x));
    }
    
    // atomicAdd(s1, (fp64)__half2float(val));
    atomicAdd(s1, (double)(val));
}

__global__ void finalize_kernel(double* s2, double a, double b, int n) {
    double h2;
    h2 = (b - a) / (2.0f * n);
    *s2 = *s2 * h2 *pi/ 3.0;
}

int main() {
    const int n = 1000000;
    double a = 0.0f;
    double b = 1.0f;
    double ss1 = 0.0;


    double* h_s1;
    hipMalloc(&h_s1, sizeof(double));
    hipMemset(h_s1, 0, sizeof(double));

  // 获取开始时间
  auto start = std::chrono::high_resolution_clock::now();


    // 计算π
    compute_pi_kernel<<<1, 1>>>();
    hipDeviceSynchronize();

    // 积分计算
    dim3 block(256);
    dim3 grid((2 * n + block.x - 1) / block.x);
    integrate_kernel<<<grid, block>>>(a, b, h_s1, n);
    
    // 最终结果计算
    finalize_kernel<<<1, 1>>>(h_s1, a, b, n);

    // 取回结果
    hipMemcpy(&ss1, h_s1, sizeof(double), hipMemcpyDeviceToHost);
  
  // 获取结束时间
  auto end = std::chrono::high_resolution_clock::now();
  // 计算并输出执行时间
  auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
  std::cout << "Execution time: " << duration.count() << " nanoseconds" << std::endl;
  double program_error=abs(abs(ss1)-abs(2.666666666666256e+00));
  printf("绝对误差为：%.15e\n",program_error);
//   printf("绝对误差为：%.15e\n",ss1);
  json data;
  // 读取现有 JSON 文件
  std::ifstream inFile("/home/zjn/mytool4/src/RuntimeAndError.json");
  if (inFile.good()) {
      try {
          data = json::parse(inFile);
      } catch (json::parse_error& e) {
          std::cerr << "JSON 解析错误，创建新文件。错误信息: " << e.what() << std::endl;
      }
  }
  inFile.close();
  // 更新当前染色体的数据
  data["chromosome1"]["runtime"] = duration.count(); // 此处键名会被 sed 动态替换
  data["chromosome1"]["error"] = program_error;
  // 写回文件
  std::ofstream file("/home/zjn/mytool4/src/RuntimeAndError.json");
  std::cout << "RuntimeAndError.json更新完成"<< std::endl;
  if (file.is_open()) {
      file << data.dump(4);
      file.close();
  } else {
      std::cerr << "无法写入文件！" << std::endl;
      return 1;
  }


    hipFree(h_s1);
    return 0;
}