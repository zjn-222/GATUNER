#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
//全Fp64
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <math.h>
#include <stdio.h>


//测量时间加的
#include <chrono>
#include <iostream>
#include <fstream>
#include <iostream>
#include <iomanip>
//json库
#include "/usr/include/nlohmann/json.hpp"
using json = nlohmann::json;


// CUDA 设备端的贝塞尔函数计算
__device__ double bessel_J0(double x1) {
    const double EPS = 1e-10;
    const int MAXIT = 10000;
    
    double sum = 0.0;
    double term = 1.0;

    for (int k = 0; fabs(term) > EPS && k < MAXIT; ++k) {
        term = pow(-1, k) * pow(x1 / 2.0, 2 * k) / (tgamma(k + 1) * tgamma(k + 1));
        sum = sum + term;
    }
    return sum;
}

// CUDA 核函数
__global__ void bessel_kernel(double* d_x, double* d_result) {
    *d_result = bessel_J0(*d_x);
}

int main() {
    double x = 1.3456;
    double result;
    
    // 分配设备内存
    double *d_x, *d_result;
    hipMalloc((void**)&d_x, sizeof(double));
    hipMalloc((void**)&d_result, sizeof(double));

    // 拷贝数据到设备
    hipMemcpy(d_x, &x, sizeof(double), hipMemcpyHostToDevice);
    
  // 获取开始时间
  auto start = std::chrono::high_resolution_clock::now();


    // 启动核函数（使用单个线程块和单个线程）
    bessel_kernel<<<1, 1>>>(d_x, d_result);

    // 等待设备计算完成
    hipDeviceSynchronize();


  // 获取结束时间
  auto end = std::chrono::high_resolution_clock::now();


    // 拷贝结果回主机
    hipMemcpy(&result, d_result, sizeof(double), hipMemcpyDeviceToHost);

    // 输出结果（添加了输出语句）
    printf("J0(%.4f) = %.15f\n", x, result);

    // 释放设备内存
    hipFree(d_x);
    hipFree(d_result);

  // 计算并输出执行时间
  auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
  std::cout << "Execution time: " << duration.count() << " nanoseconds" << std::endl;
  double program_error=abs(abs(result)-abs(0.596060578305021));
  printf("绝对误差为：%.15e\n",program_error);
  printf("结果为：%.15e\n",result);
  //并行版
  //每个chromosome个体独立写入数据到/home/zjn/mytool4/josn/chromosome1.json，最终再合并成RuntimeAndError.json
  json data;
  // 更新当前染色体的数据
  data["chromosome1"]["runtime"] = duration.count(); // 此处键名会被 sed 动态替换
  data["chromosome1"]["error"] = program_error;
  std::string json_path="/home/zjn/mytool4/json/";
  json_path+="chromosome1";
  json_path+=".json";
  // 写回文件
  std::ofstream file(json_path);
  std::cout << "RuntimeAndError.json更新完成"<< std::endl;
  if (file.is_open()) {
      file << data.dump(4);
      file.close();
  } else {
      std::cerr << "无法写入文件！" << std::endl;
      return 1;
  }

    return 0;
}