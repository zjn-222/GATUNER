#include "hip/hip_runtime.h"
//全Fp64
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <math.h>
#include <stdio.h>


//测量时间加的
#include <chrono>
#include <iostream>
#include <fstream>
#include <iostream>
#include <iomanip>
//json库
#include "/usr/include/nlohmann/json.hpp"
using json = nlohmann::json;

// typedef half fp16; // CUDA半精度类型
// typedef float fp32;
// typedef double fp64;

// 设备端全局变量
__device__ double pi;        // 存储π值
__device__ double s1_global; // 存储中间计算结果

// 设备函数
__device__ double fun(double xarg) {
    return sin(pi * xarg);
}

// 初始化π值的核函数
__global__ void compute_pi_kernel() {
    pi = acos(-1.0f);
}

// 初始化中间结果的核函数
__global__ void init_s1_kernel() {
    s1_global = 0.0;
}

// 积分计算核函数（去除了指针参数）
__global__ void integrate_kernel(double a, double b, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= 2 * n) return;

    double h1;
    h1 = (b - a) / (2.0 * n);
    double x ;
    x= idx * h1;

    double val ;
    val= (idx % 2 == 0) ? (4.0 * fun(x)) : (2.0 * fun(x));
    atomicAdd(&s1_global, val);
}

// 最终计算核函数（去除了指针参数）
__global__ void finalize_kernel(double a, double b, int n) {
    double h2;
    h2 = (b - a) / (2.0 * n);
    s1_global = s1_global * h2 * pi / 3.0;
}

int main() {
    const int n = 1000000;
    double a = 0.0;
    double b = 1.0;
    double ss1 = 0.0;

    // 初始化设备端变量
    init_s1_kernel<<<1, 1>>>();
    compute_pi_kernel<<<1, 1>>>();
    hipDeviceSynchronize();

  // 获取开始时间
  auto start = std::chrono::high_resolution_clock::now();


     // 执行积分计算
    dim3 block(256);
    dim3 grid((2 * n + block.x - 1) / block.x);
    integrate_kernel<<<grid, block>>>(a, b, n);
    
    // 执行最终计算
    finalize_kernel<<<1, 1>>>(a, b, n);
    
    // 拷贝结果回主机
    hipMemcpyFromSymbol(&ss1, HIP_SYMBOL(s1_global), sizeof(double));
  
  // 获取结束时间
  auto end = std::chrono::high_resolution_clock::now();
  // 计算并输出执行时间
  auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
  std::cout << "Execution time: " << duration.count() << " nanoseconds" << std::endl;
  double program_error=abs(abs(ss1)-abs(1.999999999999244e+00));
  printf("绝对误差为：%.15e\n",program_error);
  printf("绝对误差为：%.15e\n",ss1);
  json data;
  // 读取现有 JSON 文件
  std::ifstream inFile("/home/zjn/mytool4/src/RuntimeAndError.json");
  if (inFile.good()) {
      try {
          data = json::parse(inFile);
      } catch (json::parse_error& e) {
          std::cerr << "JSON 解析错误，创建新文件。错误信息: " << e.what() << std::endl;
      }
  }
  inFile.close();
  // 更新当前染色体的数据
  data["chromosome1"]["runtime"] = duration.count(); // 此处键名会被 sed 动态替换
  data["chromosome1"]["error"] = program_error;
  // 写回文件
  std::ofstream file("/home/zjn/mytool4/src/RuntimeAndError.json");
  std::cout << "RuntimeAndError.json更新完成"<< std::endl;
  if (file.is_open()) {
      file << data.dump(4);
      file.close();
  } else {
      std::cerr << "无法写入文件！" << std::endl;
      return 1;
  }



    return 0;
}